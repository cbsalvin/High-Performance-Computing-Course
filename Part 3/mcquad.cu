//
//  main.c
//  problem 1
//
//  Created by Alex CONG on 2/6/2016.
//  Copyright © 2016 Alex CONG. All rights reserved.
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>


__global__ void mcquad(double* x, int N, int T, long int seed)
{
hiprandState_t state;
// generate the random seed
hiprand_init(blockIdx.x, 0,0, &state);
int j;
double temp;
int tid = blockIdx.x;
if (tid<T)
{
// run the Monte-Carlo
for(j=0;j<N;j++)
{
temp=hiprand(&state) /(float)(0x0FFFFFFFFUL);
x[tid]=x[tid]+cos(-log(temp));
}
x[tid]=x[tid]/N;
}

}
int main(int argc, char* argv[])
{
int N,T;
int i=0;


// create time function
float elapsedTime;
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);

FILE* inputfile = fopen(argv[1], "r");

fscanf(inputfile, "%d", &N); // read an integer N for example
fscanf(inputfile, "%d", &T);// read time period T
fclose(inputfile);
// initial the data
double x[T];
double* dev_x;
int dev_N;
int dev_T;

dev_N=N;
dev_T=T;

hipMalloc((void**)&dev_x, T*sizeof(double));

for (i=0;i<T;i++)
{
x[i]=0.;
}

hipMemcpy(dev_x, x, T*sizeof(double), hipMemcpyHostToDevice);

mcquad<<<T,1>>>(dev_x,dev_N,dev_T,time(NULL));

hipMemcpy(x, dev_x, T*sizeof(double), hipMemcpyDeviceToHost);

// Time function
hipFree(dev_x);
hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&elapsedTime,start,stop);
printf("The Elapsed Time is %f seconds \n",elapsedTime/(float)1000);

FILE* outfile=fopen(argv[2], "w");
fwrite(x, sizeof(double), T, outfile);
fclose(outfile);
return 0;
}

